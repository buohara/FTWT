#include "hip/hip_runtime.h"
#include "nnkernels.cuh"

/**
 * [addKernel description]
 * @param c [description]
 * @param a [description]
 * @param b [description]
 */

__global__ 
void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

/**
 * [hadamard description]
 * @param a [description]
 * @param b [description]
 */

__global__
void hadamard(double *a, double *b)
{
    int i = threadIdx.x;
    a[i] = a[i] * b[i];
}

/**
 * [sigmoid description]
 * @param zVec        [description]
 * @param activations [description]
 * @param sps         [description]
 */

__global__
void sigmoid(double *zVec, double *activations, double *sps)
{
    int i = threadIdx.x;

    double sig = 1.0 / (1.0 + exp(-zVec[i]));
    activations[i] = sig;
    sps[i] = sig * (1.0 - sig);
}

/**
 * [had description]
 * @param a           [description]
 * @param b           [description]
 * @param numElements [description]
 */

void had(double *a, double *b, int numElements)
{
    hadamard <<< 1, numElements >>> (a, b);
    return;
}

/**
 * [sigmoids description]
 * @param zVec        [description]
 * @param activations [description]
 * @param sps         [description]
 * @param numElements [description]
 */

void sigmoids(double *zVec, double *activations, double *sps, int numElements)
{
    sigmoid <<< 1, numElements >>> (zVec, activations, sps);
    return;
}